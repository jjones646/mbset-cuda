/*
 * File:   MBSet.cu
 *
 * Created on November 22, 2015
 *
 * Purpose:  This program displays Mandelbrot set using the GPU via CUDA and
 * OpenGL immediate mode.
 *
 */

#include <iostream>
#include <string>
#include <sstream>
#include <stack>
#include <stdio.h>

#include <hip/hip_runtime_api.h>
#include <GL/freeglut.h>

#include "Complex.cu"

using namespace std;

// Size of window in pixels, both width and height
static const size_t WINDOW_DIM = 512;
// Msximum Iterations
static const size_t MAX_IT = 2000;
static const std::string WINDOW_BASENAME = "Mandelbrot";

// Initial screen coordinates, both host and device.
Complex minC(-2.0, -1.2);
Complex maxC(1.0, 1.8);
Complex* dev_minC;
Complex* dev_maxC;

// Define the RGB Class
class RGB
{
public:
  RGB()
    : r(0), g(0), b(0) {}
  RGB(double r0, double g0, double b0)
    : r(r0), g(g0), b(b0) {}
public:
  double r;
  double g;
  double b;
};

RGB* colors = 0; // Array of color values

void InitializeColors(void)
{
  colors = new RGB[MAX_IT + 1];
  for (size_t i = 0; i < MAX_IT; ++i)
  {
    if (i < 5)
    { // Try this.. just white for small it counts
      colors[i] = RGB(1, 1, 1);
    }
    else
    {
      colors[i] = RGB(drand48(), drand48(), drand48());
    }
  }
  colors[MAX_IT] = RGB(); // black
}

// callback for keypress - (x,y) coordinate of mouse also given for cb routine
void KeyboardCB(unsigned char key, int x, int y)
{
  cout << "Keyboard event:\tkey=" << key << "\tlocation=(" << x << "," << y << ")" << endl;
}

// callback for mouse click
void MouseCB(int button, int state, int x, int y)\
{
  cout << "Mouse event:\tbutton=" << button
  << "\tstate=" << state << "\tlocation=(" << x << "," << y << ")" << endl;
  // Possible buttons - left button is only guaranteed button to exist on a system
  // GLUT_LEFT_BUTTON = 0
  // GLUT_MIDDLE_BUTTON = 1
  // GLUT_RIGHT_BUTTON = 2
  // scroll up = 3
  // scroll down = 4
  // back click = 7
  // forward click = 8

  // Possible states due to release/press
  // GLUT_UP = 1
  // GLUT_DOWN = 0
}

// callback when the mouse moves within the window without a buttom press
void MousePassiveCB(int x, int y)
{
  char buf[12];
  sprintf(buf, "(%u,%u)", x, y);
  std::string newWinName = WINDOW_BASENAME + "\t" + buf;
  glutSetWindowTitle(newWinName.c_str());
}

// callback for display
void DisplayCB(void)
{
  // clear all
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  glMatrixMode(GL_MODELVIEW);
  // Clear the matrix
  glLoadIdentity();
  // Set the viewing transformation
  gluLookAt(0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
  glutSwapBuffers();  // for double buffering
}


int main(int argc, char** argv)
{
  // Initialize OPENGL
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);

  // get our screen dimensions & set our window name depending on what are parameters are
  const size_t display_width = glutGet(GLUT_SCREEN_WIDTH);
  const size_t display_height = glutGet(GLUT_SCREEN_HEIGHT);
  glutInitWindowSize(WINDOW_DIM, WINDOW_DIM);
  glutInitWindowPosition(100, 100);
  glutCreateWindow(WINDOW_BASENAME.c_str());
  glutSetCursor(GLUT_CURSOR_CROSSHAIR); // cause...why not?

  // Set up necessary host and device buffers

  // set up the opengl callbacks for display, mouse and keyboard
  glutDisplayFunc(DisplayCB);
  glutKeyboardFunc(KeyboardCB);
  glutMouseFunc(MouseCB);
  glutPassiveMotionFunc(MousePassiveCB);

  // Calculate the interation counts
  // Grad students, pick the colors for the 0 .. 1999 iteration count pixels


  InitializeColors();
  // This will callback the display, keyboard and mouse
  glutMainLoop();

  return 0;
}
